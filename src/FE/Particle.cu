#include "hip/hip_runtime.h"
/** ************************************************************************ 
* @brief     source file for GPU Routines of Particle Deposition within the problem
             
* @brief  Description :

            Contains all the functions that are needed to calculate the particle position within the cells.

            Functions and formulas are same as implemented in the sequential Implementation of ParMooN.
            They are just simplified ( in Datastructure wise ) for implementing in GPU

* @author  Thivin Anandh D
* @date      20-Jun-2023
* @History   Jun 20  - Creation of Cuda Routines
 ************************************************************************  */

#include <hip/hip_runtime.h>
#include "hip/hip_runtime_api.h"
#include <AllClasses.h>
#include <FEDatabase3D.h>
#include "Particle.h"
#include <cmath>

struct __device__ Vector3D_Cuda
{
    double x;
    double y;
    double z;
};

struct __device__ Pair_Cuda
{
    int first;
    int second;
};

// Struct to store the value inside BFS queue
struct __device__ value {
    int cell_no;
    int current_depth;
};

// struct to store the BFS queue
struct __device__ Node {
    value data;
    Node* next;
};

// A simple queue implementation for performing BFS
class Queue_CUDA {
public:
    __device__ Queue_CUDA() : head(nullptr), size(0) {}

    __device__ void push(int cell_no, int current_depth) {
        Node* newNode = new Node{{cell_no, current_depth}, nullptr};
        if (head == nullptr) {
            head = newNode;
        } else {
            Node* current = head;
            while (current->next != nullptr) {
                current = current->next;
            }
            current->next = newNode;
        }
        size++;
    }

    __device__ struct value pop_front() {
        if (head == nullptr) {
            return {};
        }
        Node* temp = head;
        // get the value of the current head
        struct value val = head->data;
        head = head->next;
        delete temp;
        size--;

        return val;
    }

    __device__ int getSize() const {
        return size;
    }

    __device__ void print() const {
        Node* current = head;
        while (current != nullptr) {
            printf("%d %d\n", current->data.cell_no, current->data.current_depth);
            current = current->next;
        }
    }

private:
    __device__ Node* head;
    __device__ int size;
};

__device__ struct Vector3D_Cuda Obtain_velocity_at_a_point(int current_cell,
                                            int tid,
                                           double* d_m_particle_position_x,
                                           double* d_m_particle_position_y,
                                           double* d_m_particle_position_z,
                                           double* d_m_particle_velocity_x,
                                           double* d_m_particle_velocity_y,
                                           double* d_m_particle_velocity_z,
                                           double* d_m_cell_vertices_x,
                                           double* d_m_cell_vertices_y,
                                           double* d_m_cell_vertices_z,
                                            double* d_m_velocity_nodal_values_x,
                                            double* d_m_velocity_nodal_values_y,
                                            double* d_m_velocity_nodal_values_z,
                                           int* d_m_global_dof_indices,
                                           int* d_m_begin_indices)
{
        // -- Substep 1: Perform Setcell and get the reference values from original cells
        // -- Substep 2: Perform Interpolation of velocity values at the given particle position


        // -- Substep 1: Perform Setcell and get the reference values from original cells
        double x0, x1, x2, x3;
        double y0, y1, y2, y3;
        double z0, z1, z2, z3;

        // Fill the values of the vertices of the cell
        x0 = d_m_cell_vertices_x[4 * current_cell];
        x1 = d_m_cell_vertices_x[4 * current_cell + 1];
        x2 = d_m_cell_vertices_x[4 * current_cell + 2];
        x3 = d_m_cell_vertices_x[4 * current_cell + 3];

        y0 = d_m_cell_vertices_y[4 * current_cell];
        y1 = d_m_cell_vertices_y[4 * current_cell + 1];
        y2 = d_m_cell_vertices_y[4 * current_cell + 2];
        y3 = d_m_cell_vertices_y[4 * current_cell + 3];

        z0 = d_m_cell_vertices_z[4 * current_cell];
        z1 = d_m_cell_vertices_z[4 * current_cell + 1];
        z2 = d_m_cell_vertices_z[4 * current_cell + 2];
        z3 = d_m_cell_vertices_z[4 * current_cell + 3];


        double xc0=x0;
        double xc1=x1-x0;
        double xc2=x2-x0;
        double xc3=x3-x0;

        double yc0=y0;
        double yc1=y1-y0;
        double yc2=y2-y0;
        double yc3=y3-y0;

        double zc0=z0;
        double zc1=z1-z0;
        double zc2=z2-z0;
        double zc3=z3-z0;

        double detjk= xc1*yc2*zc3 + xc2*yc3*zc1 + xc3*yc1*zc2
                -xc3*yc2*zc1 - xc2*yc1*zc3 - xc1*yc3*zc2;
        
        // Sub block 2 : Get ref values from original cells
        double X = d_m_particle_position_x[tid];
        double Y = d_m_particle_position_y[tid];
        double Z = d_m_particle_position_z[tid];

        double xt=(X - xc0)/detjk;
        double yt=(Y - yc0)/detjk;
        double zt=(Z - zc0)/detjk;

        double xi  = (yc2*zc3 - yc3*zc2)*xt - (xc2*zc3 - xc3*zc2)*yt + (xc2*yc3 - xc3*yc2)*zt;
        double eta = -(yc1*zc3 - yc3*zc1)*xt + (xc1*zc3 - xc3*zc1)*yt - (xc1*yc3 - xc3*yc1)*zt;
        double zeta = (yc1*zc2 - yc2*zc1)*xt - (xc1*zc2 - xc2*zc1)*yt + (xc1*yc2 - xc2*yc1)*zt;

        // -- Get basis function values for the given cell
        double values[10];   // Hardcoded - THIVIN - the size of basis function is hardcoded for Tetraheadral p2
        
        
        double t1 = xi*xi;
        double t2 = xi*eta;
        double t3 = xi*zeta;
        double t4 = eta*eta;
        double t5 = eta*zeta;
        double t6 = zeta*zeta;

        values[0] = 1.0-3.0*xi-3.0*eta-3.0*zeta+2.0*t1+4.0*t2+4.0*t3
                    +2.0*t4+4.0*t5+2.0*t6;
        values[1] = 4.0*xi-4.0*t1-4.0*t2-4.0*t3;
        values[2] = -xi+2.0*t1;
        values[3] = 4.0*eta-4.0*t2-4.0*t4-4.0*t5;
        values[4] = 4.0*t2;
        values[5] = -eta+2.0*t4;
        values[6] = 4.0*zeta-4.0*t3-4.0*t5-4.0*t6;
        values[7] = 4.0*t3;
        values[8] = 4.0*t5;
        values[9] = -zeta+2.0*t6;


        // Lets get the start and end indices of the global dof indices
        int start_index = d_m_begin_indices[current_cell];
        int end_index = d_m_begin_indices[current_cell+1];
        
        // Now lets get the global dof indices
        int global_dof_indices[10];
        double nodal_velocity_x[10];
        double nodal_velocity_y[10];
        double nodal_velocity_z[10];

        double interpolated_velocity_x = 0.0;
        double interpolated_velocity_y = 0.0;
        double interpolated_velocity_z = 0.0;


        for(int i=0; i<10; i++)
        {
            global_dof_indices[i] = d_m_global_dof_indices[start_index+i];
            nodal_velocity_x[i] = d_m_velocity_nodal_values_x[global_dof_indices[i]];
            nodal_velocity_y[i] = d_m_velocity_nodal_values_y[global_dof_indices[i]];
            nodal_velocity_z[i] = d_m_velocity_nodal_values_z[global_dof_indices[i]];

            interpolated_velocity_x += values[i]*nodal_velocity_x[i];
            interpolated_velocity_y += values[i]*nodal_velocity_y[i];
            interpolated_velocity_z += values[i]*nodal_velocity_z[i];

            
        }

        // Create a new struct to store the interpolated velocity values
        struct Vector3D_Cuda interpolated_velocities;    

        // Assign the interpolated velocity to the particle
        interpolated_velocities.x = interpolated_velocity_x;
        interpolated_velocities.y = interpolated_velocity_y;
        interpolated_velocities.z = interpolated_velocity_z;

        // print the values
        // printf("Particle GPU kernel 2-%d : %f %f %f\n", tid, interpolated_velocity_x, interpolated_velocity_y, interpolated_velocity_z);
        return interpolated_velocities;
}

// GPU Function which computes the cd_cc value
__device__ double cd_cc_cuda(double fluid_density,
                          double particle_diameter,
                          double fluid_velocity,
                          double particle_velocity,
                          double fluid_dynamic_viscosity,
                          double lambda
                          )
{
    double Re_Particle = fluid_density * particle_diameter * abs(fluid_velocity - particle_velocity) / fluid_dynamic_viscosity;
    double cd = (24.0 / Re_Particle) * (1.0 + 0.15 * pow(Re_Particle, 0.687)); 
    double cc = 1.0 + ((2 * lambda) / particle_diameter) * (1.257 + 0.4 * exp(-1.0 * ((1.1 * particle_diameter) / (2 * lambda))));

    return cd/cc;
}   

// Function to set the sell and return the reference values
// this function is HARD CODED for a tetrahedral cell with p2 finite element
// HARDCODED - THIVIN
__device__ struct Vector3D_Cuda SetCell_And_Return_Reference_Value_CUDA(int current_cell,
                                                                    double* d_m_cell_vertices_x,
                                                                    double* d_m_cell_vertices_y,
                                                                    double* d_m_cell_vertices_z,
                                                                    double X,
                                                                    double Y,
                                                                    double Z)
{
    // -- Substep 1: Perform Setcell and get the reference values from original cells
        double x0, x1, x2, x3;
        double y0, y1, y2, y3;
        double z0, z1, z2, z3;

        // Fill the values of the vertices of the cell
        x0 = d_m_cell_vertices_x[4 * current_cell];
        x1 = d_m_cell_vertices_x[4 * current_cell + 1];
        x2 = d_m_cell_vertices_x[4 * current_cell + 2];
        x3 = d_m_cell_vertices_x[4 * current_cell + 3];

        y0 = d_m_cell_vertices_y[4 * current_cell];
        y1 = d_m_cell_vertices_y[4 * current_cell + 1];
        y2 = d_m_cell_vertices_y[4 * current_cell + 2];
        y3 = d_m_cell_vertices_y[4 * current_cell + 3];

        z0 = d_m_cell_vertices_z[4 * current_cell];
        z1 = d_m_cell_vertices_z[4 * current_cell + 1];
        z2 = d_m_cell_vertices_z[4 * current_cell + 2];
        z3 = d_m_cell_vertices_z[4 * current_cell + 3];


        double xc0=x0;
        double xc1=x1-x0;
        double xc2=x2-x0;
        double xc3=x3-x0;

        double yc0=y0;
        double yc1=y1-y0;
        double yc2=y2-y0;
        double yc3=y3-y0;

        double zc0=z0;
        double zc1=z1-z0;
        double zc2=z2-z0;
        double zc3=z3-z0;

        double detjk= xc1*yc2*zc3 + xc2*yc3*zc1 + xc3*yc1*zc2
                -xc3*yc2*zc1 - xc2*yc1*zc3 - xc1*yc3*zc2;
        
        // Sub block 2 : Get ref values from original cells

        double xt=(X - xc0)/detjk;
        double yt=(Y - yc0)/detjk;
        double zt=(Z - zc0)/detjk;

        double xi  = (yc2*zc3 - yc3*zc2)*xt - (xc2*zc3 - xc3*zc2)*yt + (xc2*yc3 - xc3*yc2)*zt;
        double eta = -(yc1*zc3 - yc3*zc1)*xt + (xc1*zc3 - xc3*zc1)*yt - (xc1*yc3 - xc3*yc1)*zt;
        double zeta = (yc1*zc2 - yc2*zc1)*xt - (xc1*zc2 - xc2*zc1)*yt + (xc1*yc2 - xc2*yc1)*zt;

        // Create a new struct to store the xi eta zeta values
        struct Vector3D_Cuda xi_eta_zeta;

        // Assign the values
        xi_eta_zeta.x = xi;
        xi_eta_zeta.y = eta;
        xi_eta_zeta.z = zeta;

        return xi_eta_zeta;
}

// GPU function to calculate,if a point is within the given cell
// This routine is only for a tetrahedral cell with p2 Finite element
// HARDCODED - THIVIN
__device__ bool Is_Point_In_Cell_CUDA(int cellNo, 
                                    double* d_m_cell_vertices_x,
                                    double* d_m_cell_vertices_y,
                                    double* d_m_cell_vertices_z,
                                    double x,
                                    double y,
                                    double z)
{
    double xmin = 1e+8,  ymin = 1e+8, zmin = 1e+8;
    double xmax = -1e+8,  ymax = -1e+8, zmax = -1e+8;
    int i;
    bool ret = FALSE;

    double xi, eta, zeta;

    // Set cell Routine and get the reference values
    struct Vector3D_Cuda xi_eta_zeta = SetCell_And_Return_Reference_Value_CUDA(cellNo,
                                                                    d_m_cell_vertices_x,
                                                                    d_m_cell_vertices_y,
                                                                    d_m_cell_vertices_z,
                                                                    x,
                                                                    y,
                                                                    z);
    
    // Parse the values from the struct
    xi = xi_eta_zeta.x;
    eta = xi_eta_zeta.y;
    zeta = xi_eta_zeta.z;

    // Check if the reference values are within the tolerance range
    if(-1e-4 < xi && xi < 1.0001 &&
       -1e-4 < eta && eta < 1.0001 &&
       -1e-4 < zeta && zeta < 1.0001 &&
       xi + eta + zeta < 1.0001)
    {
      ret = TRUE;
    }

    return ret;
}

__global__ void Interpolate_Velocity_CUDA(  // cell Vertices
                                            double* d_m_cell_vertices_x,
                                            double* d_m_cell_vertices_y,
                                            double* d_m_cell_vertices_z,
                                            // Velocity Nodal Values
                                            double* d_m_velocity_nodal_values_x,
                                            double* d_m_velocity_nodal_values_y,
                                            double* d_m_velocity_nodal_values_z,
                                            // Particle Positions, Current & previous
                                            double* d_m_particle_position_x,
                                            double* d_m_particle_position_y,
                                            double* d_m_particle_position_z,
                                            double* d_m_particle_previous_position_x,
                                            double* d_m_particle_previous_position_y,
                                            double* d_m_particle_previous_position_z,
                                            // Particle Velocities, Current and Previous
                                            double* d_m_particle_velocity_x,
                                            double* d_m_particle_velocity_y,
                                            double* d_m_particle_velocity_z,
                                            double* d_m_particle_previous_velocity_x,
                                            double* d_m_particle_previous_velocity_y,
                                            double* d_m_particle_previous_velocity_z,
                                            // Particle & fluid Parameters
                                            double* d_m_particle_density,
                                            double* d_m_particle_diameter,
                                            double* d_m_fluid_density,
                                            double* d_m_dynamic_viscosity_fluid,
                                            double* d_m_lambda,
                                            double* d_m_gravity_x,
                                            double* d_m_gravity_y,
                                            double* d_m_gravity_z,
                                            //Computational Parameters
                                            int* d_m_current_cell,
                                            int* d_m_previous_cell,
                                            int* d_m_global_dof_indices,
                                            int* d_m_begin_indices,

                                            // FEM parameters for deposition
                                            int* d_m_is_boundary_cell,
                                            int* d_m_corner_id,
                                            int* d_m_is_boundary_dof_present,
                                            double* d_m_boundary_dof_x,
                                            double* d_m_boundary_dof_y,
                                            double* d_m_boundary_dof_z,
                                            int* d_m_joint_id,
                                            double* d_m_joint_normal_x,
                                            double* d_m_joint_normal_y,
                                            double* d_m_joint_normal_z,
                                            double* d_m_joint_coordinate_x,
                                            double* d_m_joint_coordinate_y,
                                            double* d_m_joint_coordinate_z,

                                            // STATISTICAL Parameters
                                            int* d_m_is_deposited_particle,
                                            int* d_m_is_escaped_particle,
                                            int* d_m_is_error_particle,
                                            int* d_m_is_stagnant_particle,
                                            int* d_m_is_ghost_particle,

                                            // row pointer and column indices of the adjacency matrix
                                            int* d_m_row_pointer,
                                            int* d_m_col_index,
                                            int n_cells,
                                            int n_dOF,
                                            int n_particles_released,
                                            double time_step
                                            )
{
    // Kernel code goes here
    int tid = threadIdx.x + blockIdx.x * blockDim.x;

		if (d_m_is_deposited_particle[tid])
				return;

    if(tid < n_particles_released)
    {
        // -- BLOCK 1 : INterpolate velocity values at given particle position 
        // -- Substep 1: Identify the cell in which the particle is present

        // Get the cell_id_info
        int cell_no = d_m_current_cell[tid];
        double x0 = d_m_particle_position_x[tid];
        double y0 = d_m_particle_position_y[tid];
        double z0 = d_m_particle_position_z[tid];

        int search_depth = 2;

        // if (tid == 0)
        // {
        //     printf("GPU : tid:  %d , cell_no: %d, x0:  %f, y0:  %f, z0: %f \n", tid, cell_no, x0, y0, z0);
        // }

        // Call the funciton to obtain the velocity at the given point
        struct Vector3D_Cuda interpolated_velocities = Obtain_velocity_at_a_point(cell_no,
                                    tid,
                                           d_m_particle_position_x,
                                           d_m_particle_position_y,
                                           d_m_particle_position_z,
                                           d_m_particle_velocity_x,
                                           d_m_particle_velocity_y,
                                           d_m_particle_velocity_z,
                                           d_m_cell_vertices_x,
                                           d_m_cell_vertices_y,
                                           d_m_cell_vertices_z,
                                           d_m_velocity_nodal_values_x,
                                             d_m_velocity_nodal_values_y,
                                                d_m_velocity_nodal_values_z,
                                           d_m_global_dof_indices,
                                           d_m_begin_indices);

        // USe the interpolated velocity functions to call the Calculate_Updated_Position_CUDA function 
        // This function updates the position array automatically
        // a Double can only be declated as a double * in cuda, due to malloc, so 
        // we will convert it to double value here.
        double fluid_density = *d_m_fluid_density;
        double dynamic_viscosity_fluid = *d_m_dynamic_viscosity_fluid;
        double lambda = *d_m_lambda;
        double gravity_x = *d_m_gravity_x;
        double gravity_y = *d_m_gravity_y;
        double gravity_z = *d_m_gravity_z;

        //  if (tid == 0)
        // {
        //     printf("GPU : %f %f %f\n", interpolated_velocities.x, interpolated_velocities.y, interpolated_velocities.z);
        // }

        // Calculate cd_cc
        double cd_cc_x = 0.0;
        double cd_cc_y = 0.0;
        double cd_cc_z = 0.0;

        // Assign te interpolated velocity values 
        double fluid_velocity_x = interpolated_velocities.x;
        double fluid_velocity_y = interpolated_velocities.y;
        double fluid_velocity_z = interpolated_velocities.z;


        cd_cc_x = cd_cc_cuda(fluid_density,
                            d_m_particle_diameter[tid],
                            fluid_velocity_x,
                            d_m_particle_velocity_x[tid],
                            dynamic_viscosity_fluid,
                            lambda
                            );
        
        cd_cc_y = cd_cc_cuda(fluid_density,
                                d_m_particle_diameter[tid],
                                fluid_velocity_y,
                                d_m_particle_velocity_y[tid],
                                dynamic_viscosity_fluid,
                                lambda
                                );
        
        cd_cc_z = cd_cc_cuda(fluid_density,
                                d_m_particle_diameter[tid],
                                fluid_velocity_z,
                                d_m_particle_velocity_z[tid],
                                dynamic_viscosity_fluid,
                                lambda
                                );  
        
        // // FInd the minimum cd_cc
        // double cd_cc_min = cd_cc_x;
        // if(cd_cc_y < cd_cc_min)
        //     cd_cc_min = cd_cc_y;
        // if(cd_cc_z < cd_cc_min)
        //     cd_cc_min = cd_cc_z;
        
        // // Set the cd_cc_x, cd_cc_y, cd_cc_z to cd_cc_min
        // cd_cc_x = cd_cc_min;
        // cd_cc_y = cd_cc_min;
        // cd_cc_z = cd_cc_min;

        // Calculate the RHS 
        double rhs_x = 0.0;
        double rhs_y = 0.0;
        double rhs_z = 0.0;

        double inertial_constant = (3. / 4.) * (fluid_density / d_m_particle_density[tid]) * (1 / d_m_particle_diameter[tid]);

        rhs_x = inertial_constant * cd_cc_x * abs(fluid_velocity_x - d_m_particle_velocity_x[tid]) * (fluid_velocity_x - d_m_particle_velocity_x[tid]);
        rhs_x += gravity_x * (fluid_density - d_m_particle_density[tid]) / d_m_particle_density[tid];

        rhs_y = inertial_constant * cd_cc_y * abs(fluid_velocity_y - d_m_particle_velocity_y[tid]) * (fluid_velocity_y - d_m_particle_velocity_y[tid]);
        rhs_y += gravity_y * (fluid_density - d_m_particle_density[tid]) / d_m_particle_density[tid];

        rhs_z = inertial_constant * cd_cc_z * abs(fluid_velocity_z - d_m_particle_velocity_z[tid]) * (fluid_velocity_z - d_m_particle_velocity_z[tid]);
        rhs_z += gravity_z * (fluid_density - d_m_particle_density[tid]) / d_m_particle_density[tid];

        //  if (tid == 0)
        // {
        //     printf("GPU : %f %f %f\n", rhs_x, rhs_y, rhs_z);
        // }

        // Transfer the updated particle velocity to the previous particle velocity
        d_m_particle_previous_velocity_x[tid] = d_m_particle_velocity_x[tid];
        d_m_particle_previous_velocity_y[tid] = d_m_particle_velocity_y[tid];
        d_m_particle_previous_velocity_z[tid] = d_m_particle_velocity_z[tid];

        // Compute the updated paticle velocity using forward euler
        d_m_particle_velocity_x[tid] = rhs_x * time_step + d_m_particle_velocity_x[tid];
        d_m_particle_velocity_y[tid] = rhs_y * time_step + d_m_particle_velocity_y[tid];
        d_m_particle_velocity_z[tid] = rhs_z * time_step + d_m_particle_velocity_z[tid];

        // if (tid == 0)
        // {
        //     printf("GPU : %f %f %f\n", d_m_particle_velocity_x[tid], d_m_particle_velocity_y[tid], d_m_particle_velocity_z[tid]);
        // }

        // Transfer current particle position to previous particle position
        d_m_particle_previous_position_x[tid] = d_m_particle_position_x[tid];
        d_m_particle_previous_position_y[tid] = d_m_particle_position_y[tid];
        d_m_particle_previous_position_z[tid] = d_m_particle_position_z[tid];

        // Update the particle position using RK-2
        d_m_particle_position_x[tid] = time_step * 0.5 * (d_m_particle_velocity_x[tid] + d_m_particle_previous_velocity_x[tid]) + d_m_particle_position_x[tid];
        d_m_particle_position_y[tid] = time_step * 0.5 * (d_m_particle_velocity_y[tid] + d_m_particle_previous_velocity_y[tid]) + d_m_particle_position_y[tid];
        d_m_particle_position_z[tid] = time_step * 0.5 * (d_m_particle_velocity_z[tid] + d_m_particle_previous_velocity_z[tid]) + d_m_particle_position_z[tid];
        
        // if (tid == 0)
        // {
        //     printf("GPU : %f %f %f\n", d_m_particle_previous_velocity_x[tid], d_m_particle_previous_velocity_y[tid], d_m_particle_previous_velocity_z[tid]);
        // }


        // Check the current position of the cells within the domain 
        bool inside_domain = false;

        // Check if the particle is within the current cell
        bool inside_current_cell = Is_Point_In_Cell_CUDA(cell_no,
                                                    d_m_cell_vertices_x,
                                                    d_m_cell_vertices_y,
                                                    d_m_cell_vertices_z,
                                                    d_m_particle_position_x[tid],
                                                    d_m_particle_position_y[tid],
                                                    d_m_particle_position_z[tid]);
        if(inside_current_cell)
        {
            inside_domain = true;
            // copy the current cell to previous cell
            d_m_previous_cell[tid] = d_m_current_cell[tid];
            d_m_current_cell[tid] = cell_no;
        }
        else
        {
            //Create a local array to store the cell for queue
            int queue_cell_no[10200];
            int queue_depth[10200];
            int queue_start_cursor = 0;  // Entry Condition
            int queue_end_cursor = -1; // Entry Condition

            // Add the current cell to the queue
            queue_end_cursor += 1;
            queue_cell_no[0] = cell_no;
            queue_depth[0] = 0;

            int current_depth = 0;

            while(queue_start_cursor <= queue_end_cursor && !inside_domain)
            {
                // Pop the cell from the queue
                int current_cell = queue_cell_no[queue_start_cursor];
                int current_depth = queue_depth[queue_start_cursor];
                queue_start_cursor += 1;
                // printf("GPU : tid:  %d , current_cell: %d, current_depth:  %d \n", tid, current_cell, current_depth);

                if(current_depth == search_depth)
                {
                    continue;
                }

                int start_index = d_m_row_pointer[current_cell];
                int end_index = d_m_row_pointer[current_cell+1];
 
                // for cells in current level
                for (int index = start_index; index < end_index; index++)
                {
                    // Get the neighbour cell
                    int neighbour_cell = d_m_col_index[index];
 
                    bool inside_neighbour_cell = Is_Point_In_Cell_CUDA(neighbour_cell,
                                                    d_m_cell_vertices_x,
                                                    d_m_cell_vertices_y,
                                                    d_m_cell_vertices_z,
                                                    d_m_particle_position_x[tid],
                                                    d_m_particle_position_y[tid],
                                                    d_m_particle_position_z[tid]);
                    
                    if(inside_neighbour_cell)
                    {
                        inside_domain = true;
                        // copy the current cell to previous cell
                        d_m_previous_cell[tid] = d_m_current_cell[tid];
                        d_m_current_cell[tid] = neighbour_cell;
                        break;
                    }

                    // Add the neighbour cell to the queue
                    queue_end_cursor += 1;
                    queue_cell_no[queue_end_cursor] = neighbour_cell;
                    queue_depth[queue_end_cursor] = current_depth + 1;

                    // printf("GPU : tid:  %d , neighbour_cell: %d, current_depth:  %d, queue_end_cursor: %d \n", tid, neighbour_cell, queue_depth[queue_end_cursor], queue_end_cursor);

                }
        }

        // for (int cell_id = 0; cell_id < n_cells; cell_id++)
        // {
        //     bool insideCell = Is_Point_In_Cell_CUDA(cell_id,
        //                                             d_m_cell_vertices_x,
        //                                             d_m_cell_vertices_y,
        //                                             d_m_cell_vertices_z,
        //                                             d_m_particle_position_x[tid],
        //                                             d_m_particle_position_y[tid],
        //                                             d_m_particle_position_z[tid]);
        //     if (insideCell)
        //     {
        //         inside_domain = true;
        //         // copy the current cell to previous cell
        //         d_m_previous_cell[tid] = d_m_current_cell[tid];
        //         d_m_current_cell[tid] = cell_id;
                
        //         break;
        //     }
        // }

        // If not inside the domain, then the particle is either escaped or deposited.
        if(!inside_domain)
        {
           // Find the last cell that the particle was present in 
           int cell_no = d_m_current_cell[tid];

            // Check if the cell is a boundary cell (cell with a boundary face)
            int is_boundary_cell = d_m_is_boundary_cell[cell_no];
            int index_boundary_cell = 0;
            int index_bounday_dof = 0;
            int corner_id;
            int joint_id;
            
            if(is_boundary_cell < 0) // non positive value, preferably -99999, which is used as place holder
            {
                is_boundary_cell = 0;
            }
            else
            {
                index_boundary_cell = is_boundary_cell; // assign the index
                corner_id           = d_m_corner_id[index_boundary_cell];
            }

            // Check if boundary DOF is present
            int is_boundary_dof_present = d_m_is_boundary_dof_present[tid];

            if(is_boundary_dof_present < 0) // non positive value, preferably -99999, which is used as place holder
                is_boundary_dof_present = 0;
            else
                index_bounday_dof = is_boundary_dof_present; // assign the index
            
            // If last cell not a boundary cell, check for the boundary DOF's within the cell. 
            if(!is_boundary_cell)
            {
                if(is_boundary_dof_present)  // Atleast a boundary DOF is present
                {
                    // obtain the bound dof coordinates using the index_bounday_dof index from d_m_boundary_dof_x, d_m_boundary_dof_y, d_m_boundary_dof_z
                    double boundary_dof_x = d_m_boundary_dof_x[index_bounday_dof];
                    double boundary_dof_y = d_m_boundary_dof_y[index_bounday_dof];
                    double boundary_dof_z = d_m_boundary_dof_z[index_bounday_dof];

                    // Assign it to the particle position
                    d_m_particle_position_x[tid] = boundary_dof_x;
                    d_m_particle_position_y[tid] = boundary_dof_y;
                    d_m_particle_position_z[tid] = boundary_dof_z;

                    // Mark the particle as deposited
                    d_m_is_deposited_particle[tid] = 1;
                    return;
                }

                // FUTURE TODO : May be add a check to see if the neighbouring cells have boundary DOF's or boundary faces
                else  // Does not have a Boundary face nor a boundary DOF
                {
										// Mark the particle as deposited , for book keeping purposes
										d_m_is_deposited_particle[tid] = 1;

										// Mark the particle as error 
										d_m_is_error_particle[tid] = 1;
                    return;
                }
            }

            // if the corner id = 21, then it means, the cell has boundary faces on wall and outflow boundary
            // So the particle is at the the last cell near the exit
            // since the particle has reached here, we will consider the particle escaped via the outflow boundary
            if (is_boundary_cell && corner_id == 21 ) 
            {
                // Mark the particle as deposited , for book keeping purposes
                d_m_is_deposited_particle[tid] = 1;

                // Mark the particle as escaped
                d_m_is_escaped_particle[tid] = 1;

                return;
            }

             // Check if the particle is from a corner shared by two bdids 2 and 0
             // this is an inlet between wall and inlet, so we will consider the particle escaped via the wall
            if (is_boundary_cell && corner_id == 20)
            {
                joint_id  = d_m_joint_id[index_boundary_cell];  
            }

            // If a paricle escaped from a cell, in which there is only inlet surface then its an error particle
            // This will only happen if there is a backflow
            if(corner_id == 0)
            {
                // Mark the particle as deposited , for book keeping purposes
                d_m_is_deposited_particle[tid] = 1;
                
                // Mark the particle as error 
                d_m_is_error_particle[tid] = 1;
                
                return;
            }
            else // Corner ID is 1 or 2
            {   
                // get the joint id
                joint_id  = d_m_joint_id[index_boundary_cell];
            }

            // With the obtained joint id and get the corresponding joint normal and joint coordinate
            double joint_normal_x = d_m_joint_normal_x[index_boundary_cell];
            double joint_normal_y = d_m_joint_normal_y[index_boundary_cell];
            double joint_normal_z = d_m_joint_normal_z[index_boundary_cell];

            double joint_coordinate_x = d_m_joint_coordinate_x[index_boundary_cell];
            double joint_coordinate_y = d_m_joint_coordinate_y[index_boundary_cell];
            double joint_coordinate_z = d_m_joint_coordinate_z[index_boundary_cell];

            // Get the structs
            struct Vector3D_Cuda first_position;
            struct Vector3D_Cuda second_position;
            struct Vector3D_Cuda line_vector;
            struct Vector3D_Cuda point_on_surface;
            struct Vector3D_Cuda normal_surface;
            struct Vector3D_Cuda temp1;
            struct Vector3D_Cuda temp2;

            first_position.x = d_m_particle_previous_position_x[tid];
            first_position.y = d_m_particle_previous_position_y[tid];
            first_position.z = d_m_particle_previous_position_z[tid];

            second_position.x = d_m_particle_position_x[tid];
            second_position.y = d_m_particle_position_y[tid];
            second_position.z = d_m_particle_position_z[tid];

            normal_surface.x = joint_normal_x;
            normal_surface.y = joint_normal_y;
            normal_surface.z = joint_normal_z;

            point_on_surface.x = joint_coordinate_x;
            point_on_surface.y = joint_coordinate_y;
            point_on_surface.z = joint_coordinate_z;

            //  u = p1 - p0
            line_vector.x = first_position.x - second_position.x;
            line_vector.y = first_position.y - second_position.y;
            line_vector.z = first_position.z - second_position.z;

            // Dot
            double dot = normal_surface.x * first_position.x + normal_surface.y * first_position.y + normal_surface.z * first_position.z;

            if(fabs(dot - 0.0) > 1e-3)
            {
                // w = p0 - pC0
                temp1.x = first_position.x - point_on_surface.x;
                temp1.y = first_position.y - point_on_surface.y;
                temp1.z = first_position.z - point_on_surface.z;

                double fac = -1.0 * (normal_surface.x * temp1.x + normal_surface.y * temp1.y + normal_surface.z * temp1.z);
                fac /= fac;

                // u = u*fac
                line_vector.x = line_vector.x * fac;
                line_vector.y = line_vector.y * fac;
                line_vector.z = line_vector.z * fac;

                temp2.x = line_vector.x + first_position.x;
                temp2.y = line_vector.y + first_position.y;
                temp2.z = line_vector.z + first_position.z;

                // Mark the Particle as deposited
                d_m_is_deposited_particle[tid] = 1;

                d_m_particle_position_x[tid] = temp2.x;
                d_m_particle_position_y[tid] = temp2.y;
                d_m_particle_position_z[tid] = temp2.z;

                // if bdId/corner ID is 1, then mark the particle as escaped
                if(corner_id == 1)
                {
                    // Mark the Particle as deposited
                    d_m_is_deposited_particle[tid] = 1;

                    // Mark the particle as escaped
                    d_m_is_escaped_particle[tid] = 1;

                    return;
                }
            }
            else
            {
                if(corner_id ==1)
                {
                    d_m_is_deposited_particle[tid] = 1; // Mark the Particle as deposited
                    d_m_is_escaped_particle[tid] = 1;   // Mark the particle as escaped
                }
                // mark them as deposited 
                d_m_is_deposited_particle[tid] = 1;

                // Ghost particle, make the vertex position as deposition position
                d_m_particle_position_x[tid] = point_on_surface.x;
                d_m_particle_position_y[tid] = point_on_surface.y;
                d_m_particle_position_z[tid] = point_on_surface.z;

                d_m_is_ghost_particle[tid] = 1;
                
            }

        }

    }

}

__global__ void DetectStagnantParticles_CUDA(double* d_m_particle_position_x,
                                             double* d_m_particle_position_y,
                                             double* d_m_particle_position_z,
                                             double* d_m_particle_stagnant_position_x,
                                             double* d_m_particle_stagnant_position_y,
                                             double* d_m_particle_stagnant_position_z,
                                             int* d_m_is_stagnant_particle,
                                             int* d_m_is_deposited_particle)
{
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
		if (d_m_is_deposited_particle[tid])
				return;

		double distance = sqrt(pow(d_m_particle_position_x[tid] - d_m_particle_stagnant_position_x[tid], 2) +
													 pow(d_m_particle_position_y[tid] - d_m_particle_stagnant_position_y[tid], 2) +
													 pow(d_m_particle_position_z[tid] - d_m_particle_stagnant_position_z[tid], 2));
		d_m_particle_stagnant_position_x[tid] = d_m_particle_position_x[tid];
		d_m_particle_stagnant_position_y[tid] = d_m_particle_position_y[tid];
		d_m_particle_stagnant_position_z[tid] = d_m_particle_position_z[tid];

		if (distance >= 0.0001) 
			return;

		d_m_is_stagnant_particle[tid] = 1;
		d_m_is_deposited_particle[tid] = 1;
}

void TParticles::SetupCudaDataStructures(TFESpace3D* fespace)
{
    // get the collection of cells from the fespace
    TCollection *coll = fespace->GetCollection();

    // get No of cells in the collection
    int N_Cells = coll->GetN_Cells();

    // get N_dof
    int N_DOF = fespace->GetN_DegreesOfFreedom();

    // get the global indices and begin index of the cells from fespace
    int *GlobalNumbers = fespace->GetGlobalNumbers();
    int *BeginIndex = fespace->GetBeginIndex();

    // get the last value in the begin index array
    int size_of_global_numbers = BeginIndex[N_Cells];

    // Allocate memory for the cell vertices
    h_m_cell_vertices_x = new double[4 * N_Cells];
    h_m_cell_vertices_y = new double[4 * N_Cells];
    h_m_cell_vertices_z = new double[4 * N_Cells];

    // Allocate memory for global indices of the cells
    h_m_global_dof_indices = new int[size_of_global_numbers];

    // Allocate memory for the begin index of the cells
    h_m_begin_indices  = new int[N_Cells + 1];

    // allocate memory for velocity values
    h_m_velocityX = new double[N_DOF];
    h_m_velocityY = new double[N_DOF];
    h_m_velocityZ = new double[N_DOF];


    // -- Now lets fill these values, so that these can be copied to the GPU

    // fill the cell vertices
    for(int i = 0; i < N_Cells; i++){
        // get the cell
        TBaseCell *cell = coll->GetCell(i);
        
        // fill the vertices
        for(int j = 0; j < 4; j++){
            double x0,y0,z0;
            cell->GetVertex(j)->GetCoords(x0, y0, z0);
            h_m_cell_vertices_x[4 * i + j] = x0;
            h_m_cell_vertices_y[4 * i + j] = y0;
            h_m_cell_vertices_z[4 * i + j] = z0;
        }
    }

    // Lets fill the global indices and begin index
    for(int i = 0; i < size_of_global_numbers; i++){
        h_m_global_dof_indices[i] = GlobalNumbers[i];
    }

    for(int i = 0; i < N_Cells + 1; i++){
        h_m_begin_indices[i] = BeginIndex[i];
    }

    // Refer to Documentation "DepositionLogic.md" for further details. 
    // In a Nutshell, there will be a `is_***` array, which will be of size N_Cells
    // if the cell does not match the category, then the value will be given as `-99999`
    // if the cell do match the category, then the value will be given as a positive number, which will be the index at which the corresponding values will be present

    // For Eg : Lets say , there are 5 cells , and cells 0,2,3 has boundary_faces, then the is_boundary_faces array will be as follows
    // is_boundary_faces = [0, -99999,1, 2, -99999], and the joint ids of the cells 0,2,3 wil be saved in the array
    // joint_ids = [4, 0, 1] , where 4 is the joint id of cell 0, 0 is the joint id of cell 2 and 1 is the joint id of cell 3

    // Lets fill the is_boundary_faces array
    h_m_is_boundary_cell = new int[N_Cells];

    // get size of the m_mapBoundaryFaceIds map
    int size_of_boundary_face = m_mapBoundaryFaceIds.size();
    
    // create joint_ids array and corner_ids array of size equal to the size of the m_mapBoundaryFaceIds map
    h_m_joint_id = new int[size_of_boundary_face];
    h_m_corner_id = new int[size_of_boundary_face];

    // Allocate memory for joint normals and joint coordinates
    h_m_joint_normal_x = new double[size_of_boundary_face];
    h_m_joint_normal_y = new double[size_of_boundary_face];
    h_m_joint_normal_z = new double[size_of_boundary_face];

    h_m_joint_coordinate_x = new double[size_of_boundary_face];
    h_m_joint_coordinate_y = new double[size_of_boundary_face];
    h_m_joint_coordinate_z = new double[size_of_boundary_face];


    // Initialise the h_m_is_boundary_faces, h_m_joint_ids and h_m_corner_ids array 
    // using maps, m_mapBoundaryFaceIds, m_mapJointIds and m_mapCornerIds
    // if the key is present in the map, then the value will be the index at which the corresponding values will be present
    // if the key is not present in the map, then the value will be -99999

    int index_for_boundary_face = 0;
    // loop through the cells
    for (int cell_no = 0 ; cell_no < N_Cells; cell_no++)
    {
        // check if the cell no is present in the map
        if (m_mapBoundaryFaceIds.find(cell_no) != m_mapBoundaryFaceIds.end())
        {
            // if the cell no is present in the map, then the value will be the index at which the corresponding values will be present
            h_m_is_boundary_cell[cell_no] = index_for_boundary_face;

            // get the joint id and corner id from the map
            int joint_id = m_jointidOfBoundCells[cell_no];
            int corner_id = m_cornerTypeOfBoundCells[cell_no];

            // save the joint id and corner id in the corresponding arrays
            h_m_joint_id[index_for_boundary_face] = joint_id;
            h_m_corner_id[index_for_boundary_face] = corner_id;

            // get the joint normal and joint coordinate from the map
            double joint_normal_x ;
            double joint_normal_y ;
            double joint_normal_z ;

            double joint_coordinate_x;
            double joint_coordinate_y;
            double joint_coordinate_z;


            // Variables required for normal Computation 
            int MaxLen;
            int N_Joints;
            const int *TmpLen;
            const int *TmpFV;


            TBaseCell *cell = fespace->GetCollection()->GetCell(cell_no);
            cell->GetShapeDesc()->GetFaceVertex(TmpFV, TmpLen, MaxLen);
            TJoint *Joint = cell->GetJoint(joint_id);
            double x1, x2, x3, y1, y2, y3, z1, z2, z3;

            // Get the coordinates of the joint
            cell->GetVertex(TmpFV[joint_id * MaxLen + 0])->GetCoords(x1, y1, z1);
            cell->GetVertex(TmpFV[joint_id * MaxLen + 1])->GetCoords(x2, y2, z2);
            double t11 = x2 - x1;
            double t12 = y2 - y1;
            double t13 = z2 - z1;
            double len = sqrt(t11 * t11 + t12 * t12 + t13 * t13);
            t11 /= len;
            t12 /= len;
            t13 /= len;

            cell->GetVertex(TmpFV[joint_id * MaxLen + (TmpLen[joint_id] - 1)])->GetCoords(x2, y2, z2);
            double t21 = x2 - x1;
            double t22 = y2 - y1;
            double t23 = z2 - z1;
            len = sqrt(t21 * t21 + t22 * t22 + t23 * t23);
            t21 /= len;
            t22 /= len;
            t23 /= len;

            double N1 = t12 * t23 - t13 * t22;
            double N2 = t13 * t21 - t11 * t23;
            double N3 = t11 * t22 - t12 * t21;
            len = sqrt(N1 * N1 + N2 * N2 + N3 * N3);
            N1 /= len;
            N2 /= len;
            N3 /= len;

            // Assign the joint normal 
            joint_normal_x = N1;
            joint_normal_y = N2;
            joint_normal_z = N3;

            // Assign joint coordinate
            joint_coordinate_x = x1;
            joint_coordinate_y = y1;
            joint_coordinate_z = z1;

            // save the joint normal and joint coordinate in the corresponding arrays
            h_m_joint_normal_x[index_for_boundary_face] = joint_normal_x;
            h_m_joint_normal_y[index_for_boundary_face] = joint_normal_y;
            h_m_joint_normal_z[index_for_boundary_face] = joint_normal_z;

            h_m_joint_coordinate_x[index_for_boundary_face] = joint_coordinate_x;
            h_m_joint_coordinate_y[index_for_boundary_face] = joint_coordinate_y;
            h_m_joint_coordinate_z[index_for_boundary_face] = joint_coordinate_z;

            // increment the index_for_boundary_face
            index_for_boundary_face++;
        }
        else
        {
            // if the cell no is not present in the map, then the value will be -99999
            h_m_is_boundary_cell[cell_no] = -99999;
        }
    }


    // fill the is_boundary_dof array 
    h_m_is_boundary_dof_present = new int[N_Cells];

    // get the size of the m_mapBoundaryDofIds map
    int size_of_boundary_dof = m_BoundaryDOFsOnCell.size();

    // Allocate memory for the arrays
    h_m_boundary_dof_x = new double[size_of_boundary_dof];
    h_m_boundary_dof_y = new double[size_of_boundary_dof];
    h_m_boundary_dof_z = new double[size_of_boundary_dof];
    
    // Initialise the h_m_is_boundary_dof, h_m_joint_normal_x, h_m_joint_normal_y, h_m_joint_normal_z, h_m_joint_coordinate_x, h_m_joint_coordinate_y, h_m_joint_coordinate_z array
    // using maps, m_mapBoundaryDofIds, and compute the joint normals and joint coordinates
    
    // print size of boundary dof
    cout << "[INFORMATION] size_of_boundary_dof : " << size_of_boundary_dof << endl;

    // print num of cells
    cout << "[INFORMATION] N_Cells : " << N_Cells << endl;


    int index_for_boundary_dof = 0;
    
    for (int cell_no = 0 ; cell_no < N_Cells ; cell_no ++)
    {
        // check if the cell no is present in the map
        if (m_BoundaryDOFsOnCell.find(cell_no) != m_BoundaryDOFsOnCell.end())
        {
            // if the cell no is present in the map, then the value will be the index at which the corresponding values will be present
            h_m_is_boundary_dof_present[cell_no] = index_for_boundary_dof;
            
            std::vector<double> boundary_dof = m_BoundaryDOFsOnCell[cell_no];
            h_m_boundary_dof_x[index_for_boundary_dof] = boundary_dof[0]; // x dof
            h_m_boundary_dof_y[index_for_boundary_dof] = boundary_dof[1]; // y dof
            h_m_boundary_dof_z[index_for_boundary_dof] = boundary_dof[2]; // z dof

            // increment the index_for_boundary_dof
            index_for_boundary_dof++;
        }
        else
        {
            // if the cell no is not present in the map, then the value will be -99999
            h_m_is_boundary_dof_present[cell_no] = -99999;
        }
    }

    
   
    // ----- ALLOCATE MEMORY IN GPU FOR ALL THE DEVICE VARIABLES -----

    // Allocate memory for the cell vertices
    checkCudaErrors(hipMalloc((void**)&d_m_cell_vertices_x, 4 * N_Cells * sizeof(double)));
    checkCudaErrors(hipMalloc((void**)&d_m_cell_vertices_y, 4 * N_Cells * sizeof(double)));
    checkCudaErrors(hipMalloc((void**)&d_m_cell_vertices_z, 4 * N_Cells * sizeof(double)));

    // Allocate memory for global indices of the cells
    checkCudaErrors(hipMalloc((void**)&d_m_global_dof_indices, size_of_global_numbers * sizeof(int)));

    // Allocate memory for the begin index of the cells
    checkCudaErrors(hipMalloc((void**)&d_m_begin_indices, (N_Cells + 1) * sizeof(int)));

    // allocate memory for velocity values
    checkCudaErrors(hipMalloc((void**)&d_m_velocity_nodal_values_x, N_DOF * sizeof(double)));
    checkCudaErrors(hipMalloc((void**)&d_m_velocity_nodal_values_y, N_DOF * sizeof(double)));
    checkCudaErrors(hipMalloc((void**)&d_m_velocity_nodal_values_z, N_DOF * sizeof(double)));

    // Allocate memory for current cell and previous cell
    checkCudaErrors(hipMalloc((void**)&d_m_current_cell, N_Particles * sizeof(int)));
    checkCudaErrors(hipMalloc((void**)&d_m_previous_cell, N_Particles * sizeof(int)));

    // Allocate memory for the particle position
    checkCudaErrors(hipMalloc((void**)&d_m_particle_position_x, N_Particles * sizeof(double)));
    checkCudaErrors(hipMalloc((void**)&d_m_particle_position_y, N_Particles * sizeof(double)));
    checkCudaErrors(hipMalloc((void**)&d_m_particle_position_z, N_Particles * sizeof(double)));

    // Allocate memory for the particle previous position
    checkCudaErrors(hipMalloc((void**)&d_m_particle_previous_position_x, N_Particles * sizeof(double)));
    checkCudaErrors(hipMalloc((void**)&d_m_particle_previous_position_y, N_Particles * sizeof(double)));
    checkCudaErrors(hipMalloc((void**)&d_m_particle_previous_position_z, N_Particles * sizeof(double)));

    // Allocate memory for the particle previous position (for stagnancy check)
    checkCudaErrors(hipMalloc((void**)&d_m_particle_stagnant_position_x, N_Particles * sizeof(double)));
    checkCudaErrors(hipMalloc((void**)&d_m_particle_stagnant_position_y, N_Particles * sizeof(double)));
    checkCudaErrors(hipMalloc((void**)&d_m_particle_stagnant_position_z, N_Particles * sizeof(double)));

    // Allocate memory for the particle velocity
    checkCudaErrors(hipMalloc((void**)&d_m_particle_velocity_x, N_Particles * sizeof(double)));
    checkCudaErrors(hipMalloc((void**)&d_m_particle_velocity_y, N_Particles * sizeof(double)));
    checkCudaErrors(hipMalloc((void**)&d_m_particle_velocity_z, N_Particles * sizeof(double)));

    // Allocate memory for the particle previous velocity
    checkCudaErrors(hipMalloc((void**)&d_m_particle_previous_velocity_x, N_Particles * sizeof(double)));
    checkCudaErrors(hipMalloc((void**)&d_m_particle_previous_velocity_y, N_Particles * sizeof(double)));
    checkCudaErrors(hipMalloc((void**)&d_m_particle_previous_velocity_z, N_Particles * sizeof(double)));

    // -- Copy the data from host to device -- //

    // copy the cell vertices
    checkCudaErrors(hipMemcpy(d_m_cell_vertices_x, h_m_cell_vertices_x, 4 * N_Cells * sizeof(double), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_m_cell_vertices_y, h_m_cell_vertices_y, 4 * N_Cells * sizeof(double), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_m_cell_vertices_z, h_m_cell_vertices_z, 4 * N_Cells * sizeof(double), hipMemcpyHostToDevice));

    // copy the global indices and begin index
    checkCudaErrors(hipMemcpy(d_m_global_dof_indices, h_m_global_dof_indices, size_of_global_numbers * sizeof(int), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_m_begin_indices, h_m_begin_indices, (N_Cells + 1) * sizeof(int), hipMemcpyHostToDevice));

    // Copy the Initial particle position ( this is copied directly from the class variable, no parallel host array is created)
    checkCudaErrors(hipMemcpy(d_m_particle_position_x, position_X.data(), N_Particles * sizeof(double), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_m_particle_position_y, position_Y.data(), N_Particles * sizeof(double), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_m_particle_position_z, position_Z.data(), N_Particles * sizeof(double), hipMemcpyHostToDevice));


    // Copy the previous particle position
    checkCudaErrors(hipMemcpy(d_m_particle_previous_position_x, position_X_old.data(), N_Particles * sizeof(double), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_m_particle_previous_position_y, position_Y_old.data(), N_Particles * sizeof(double), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_m_particle_previous_position_z, position_Z_old.data(), N_Particles * sizeof(double), hipMemcpyHostToDevice));

    // Copy the previous particle position (for stagnancy check)
    checkCudaErrors(hipMemcpy(d_m_particle_stagnant_position_x, previousPosition_X.data(), N_Particles * sizeof(double), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_m_particle_stagnant_position_y, previousPosition_Y.data(), N_Particles * sizeof(double), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_m_particle_stagnant_position_z, previousPosition_Z.data(), N_Particles * sizeof(double), hipMemcpyHostToDevice));


    // Copy the particle velocity ( This will copy only zero values ), actual values will be copied in the main loop at each time step
    checkCudaErrors(hipMemcpy(d_m_particle_velocity_x, velocityX.data(), N_Particles * sizeof(double), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_m_particle_velocity_y, velocityY.data(), N_Particles * sizeof(double), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_m_particle_velocity_z, velocityZ.data(), N_Particles * sizeof(double), hipMemcpyHostToDevice));


    // Copy the previous particle velocity
    checkCudaErrors(hipMemcpy(d_m_particle_previous_velocity_x, velocityX_old.data(), N_Particles * sizeof(double), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_m_particle_previous_velocity_y, velocityY_old.data(), N_Particles * sizeof(double), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_m_particle_previous_velocity_z, velocityZ_old.data(), N_Particles * sizeof(double), hipMemcpyHostToDevice));

    // Copy the current cell and previous cell
    checkCudaErrors(hipMemcpy(d_m_current_cell, currentCell.data(), N_Particles * sizeof(int), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_m_previous_cell, previousCell.data(), N_Particles * sizeof(int), hipMemcpyHostToDevice));


    // ----- PARTICLE AND FLUID PARAMETERS ----- //
    
    // Allocate memory for particle density
    checkCudaErrors(hipMalloc((void**)&d_m_particle_density, N_Particles * sizeof(double)));

    // Allocate memory for particle diameter
    checkCudaErrors(hipMalloc((void**)&d_m_particle_diameter, N_Particles * sizeof(double)));

    // Allocate memory for fluid density : Data type is scalar and double
    checkCudaErrors(hipMalloc((void**)&d_m_fluid_density, sizeof(double)));

    // Allocate memory for fluid viscosity : Data type is scalar and double
    checkCudaErrors(hipMalloc((void**)&d_m_dynamic_viscosity_fluid, sizeof(double)));

    // Allocate memory for fluid gravity in x direction : Data type is scalar and double
    checkCudaErrors(hipMalloc((void**)&d_m_gravity_x, sizeof(double)));

    // Allocate memory for fluid gravity in y direction : Data type is scalar and double
    checkCudaErrors(hipMalloc((void**)&d_m_gravity_y, sizeof(double)));

    // Allocate memory for fluid gravity in z direction : Data type is scalar and double
    checkCudaErrors(hipMalloc((void**)&d_m_gravity_z, sizeof(double)));

    // Allocate lambda : Data type is scalar and double
    checkCudaErrors(hipMalloc((void**)&d_m_lambda, sizeof(double)));

    // Send the fluid and particle parameters to GPU, 
    // d_m_ is the device variable with dtype as double* and m_ is the host variable with dtype as std::vector<double> 
    checkCudaErrors(hipMemcpy(d_m_particle_density, m_particle_density.data(), N_Particles * sizeof(double), hipMemcpyHostToDevice));

    // Send the particle diameter
    checkCudaErrors(hipMemcpy(d_m_particle_diameter, m_particle_diameter.data(), N_Particles * sizeof(double), hipMemcpyHostToDevice));

    // Send the fluid density
    checkCudaErrors(hipMemcpy(d_m_fluid_density, &m_fluid_density, sizeof(double), hipMemcpyHostToDevice));

    // Send the fluid viscosity
    checkCudaErrors(hipMemcpy(d_m_dynamic_viscosity_fluid, &m_fluid_dynamic_viscosity, sizeof(double), hipMemcpyHostToDevice));

    // Send the fluid gravity in x direction
    checkCudaErrors(hipMemcpy(d_m_gravity_x, &m_gravity_x, sizeof(double), hipMemcpyHostToDevice));

    // Send the fluid gravity in y direction
    checkCudaErrors(hipMemcpy(d_m_gravity_y, &m_gravity_y, sizeof(double), hipMemcpyHostToDevice));

    // Send the fluid gravity in z direction
    checkCudaErrors(hipMemcpy(d_m_gravity_z, &m_gravity_z, sizeof(double), hipMemcpyHostToDevice));

    // Send the lambda
    checkCudaErrors(hipMemcpy(d_m_lambda, &m_lambda, sizeof(double), hipMemcpyHostToDevice));

    // -- TIME PARAMETERS -- //

    // Allocate memory for the time step
    checkCudaErrors(hipMalloc((void**)&d_m_time_step, sizeof(double)));

    // transfer the time step to the GPU
    checkCudaErrors(hipMemcpy(d_m_time_step, &h_m_time_step, sizeof(double), hipMemcpyHostToDevice));


    // ---- DEPOSITION PARAMETERS -- // 
    // Allocate memory for d_m_is_boundary_cell with size N_Cells and data type int
    checkCudaErrors(hipMalloc((void**)&d_m_is_boundary_cell, N_Cells * sizeof(int)));

    // Allocate memory for d_m_corner_id with size equal to the size of the m_mapBoundaryFaceIds map and data type int
    checkCudaErrors(hipMalloc((void**)&d_m_corner_id, size_of_boundary_face * sizeof(int)));

    // Allocate memory for d_m_joint_id with size equal to the size of the m_mapBoundaryFaceIds map and data type int
    checkCudaErrors(hipMalloc((void**)&d_m_joint_id, size_of_boundary_face * sizeof(int)));

    // Allocate memory for d_m_joint_normal_x with size equal to the size of the m_mapBoundaryFaceIds map and data type double
    checkCudaErrors(hipMalloc((void**)&d_m_joint_normal_x, size_of_boundary_face * sizeof(double)));

    // Allocate memory for d_m_joint_normal_y with size equal to the size of the m_mapBoundaryFaceIds map and data type double
    checkCudaErrors(hipMalloc((void**)&d_m_joint_normal_y, size_of_boundary_face * sizeof(double)));

    // Allocate memory for d_m_joint_normal_z with size equal to the size of the m_mapBoundaryFaceIds map and data type double
    checkCudaErrors(hipMalloc((void**)&d_m_joint_normal_z, size_of_boundary_face * sizeof(double)));

    // Allocate memory for d_m_joint_coordinate_x with size equal to the size of the m_mapBoundaryFaceIds map and data type double
    checkCudaErrors(hipMalloc((void**)&d_m_joint_coordinate_x, size_of_boundary_face * sizeof(double)));

    // Allocate memory for d_m_joint_coordinate_y with size equal to the size of the m_mapBoundaryFaceIds map and data type double
    checkCudaErrors(hipMalloc((void**)&d_m_joint_coordinate_y, size_of_boundary_face * sizeof(double)));

    // Allocate memory for d_m_joint_coordinate_z with size equal to the size of the m_mapBoundaryFaceIds map and data type double
    checkCudaErrors(hipMalloc((void**)&d_m_joint_coordinate_z, size_of_boundary_face * sizeof(double)));

    // Allocate memory for d_m_is_boundary_dof_present with size N_cells and data type int
    checkCudaErrors(hipMalloc((void**)&d_m_is_boundary_dof_present, N_Cells * sizeof(int)));

    // Allocate memory for d_m_boundary_dof_x with size equal to the size of the m_mapBoundaryDofIds map and data type int
    checkCudaErrors(hipMalloc((void**)&d_m_boundary_dof_x, size_of_boundary_dof * sizeof(int)));

    // Allocate memory for d_m_boundary_dof_y with size equal to the size of the m_mapBoundaryDofIds map and data type int
    checkCudaErrors(hipMalloc((void**)&d_m_boundary_dof_y, size_of_boundary_dof * sizeof(int)));

    // Allocate memory for d_m_boundary_dof_z with size equal to the size of the m_mapBoundaryDofIds map and data type int
    checkCudaErrors(hipMalloc((void**)&d_m_boundary_dof_z, size_of_boundary_dof * sizeof(int)));
    

    // Copy the values from host to these arrays 
    // copy for d_m_is_boundary_cell from h_m_is_boundary_cell with size N_Cells and data type int
    checkCudaErrors(hipMemcpy(d_m_is_boundary_cell, h_m_is_boundary_cell, N_Cells * sizeof(int), hipMemcpyHostToDevice));

    // copy for d_m_corner_id from h_m_corner_id with size equal to the size of the m_mapBoundaryFaceIds map and data type int
    checkCudaErrors(hipMemcpy(d_m_corner_id, h_m_corner_id, size_of_boundary_face * sizeof(int), hipMemcpyHostToDevice));

    // copy for d_m_joint_id from h_m_joint_id with size equal to the size of the m_mapBoundaryFaceIds map and data type int
    checkCudaErrors(hipMemcpy(d_m_joint_id, h_m_joint_id, size_of_boundary_face * sizeof(int), hipMemcpyHostToDevice));

    // copy for d_m_joint_normal_x from h_m_joint_normal_x with size equal to the size of the m_mapBoundaryFaceIds map and data type double
    checkCudaErrors(hipMemcpy(d_m_joint_normal_x, h_m_joint_normal_x, size_of_boundary_face * sizeof(double), hipMemcpyHostToDevice));

    // copy for d_m_joint_normal_y from h_m_joint_normal_y with size equal to the size of the m_mapBoundaryFaceIds map and data type double
    checkCudaErrors(hipMemcpy(d_m_joint_normal_y, h_m_joint_normal_y, size_of_boundary_face * sizeof(double), hipMemcpyHostToDevice));

    // copy for d_m_joint_normal_z from h_m_joint_normal_z with size equal to the size of the m_mapBoundaryFaceIds map and data type double
    checkCudaErrors(hipMemcpy(d_m_joint_normal_z, h_m_joint_normal_z, size_of_boundary_face * sizeof(double), hipMemcpyHostToDevice));

    // copy for d_m_joint_coordinate_x from h_m_joint_coordinate_x with size equal to the size of the m_mapBoundaryFaceIds map and data type double
    checkCudaErrors(hipMemcpy(d_m_joint_coordinate_x, h_m_joint_coordinate_x, size_of_boundary_face * sizeof(double), hipMemcpyHostToDevice));

    // copy for d_m_joint_coordinate_y from h_m_joint_coordinate_y with size equal to the size of the m_mapBoundaryFaceIds map and data type double
    checkCudaErrors(hipMemcpy(d_m_joint_coordinate_y, h_m_joint_coordinate_y, size_of_boundary_face * sizeof(double), hipMemcpyHostToDevice));

    // copy for d_m_joint_coordinate_z from h_m_joint_coordinate_z with size equal to the size of the m_mapBoundaryFaceIds map and data type double
    checkCudaErrors(hipMemcpy(d_m_joint_coordinate_z, h_m_joint_coordinate_z, size_of_boundary_face * sizeof(double), hipMemcpyHostToDevice));

    // copy for d_m_is_boundary_dof_present from h_m_is_boundary_dof_present with size N_cells and data type int
    checkCudaErrors(hipMemcpy(d_m_is_boundary_dof_present, h_m_is_boundary_dof_present, N_Cells * sizeof(int), hipMemcpyHostToDevice));

    // copy for d_m_boundary_dof_x from h_m_boundary_dof_x with size equal to the size of the m_mapBoundaryDofIds map and data type int
    checkCudaErrors(hipMemcpy(d_m_boundary_dof_x, h_m_boundary_dof_x, size_of_boundary_dof * sizeof(int), hipMemcpyHostToDevice));

    // copy for d_m_boundary_dof_y from h_m_boundary_dof_y with size equal to the size of the m_mapBoundaryDofIds map and data type int
    checkCudaErrors(hipMemcpy(d_m_boundary_dof_y, h_m_boundary_dof_y, size_of_boundary_dof * sizeof(int), hipMemcpyHostToDevice));

    // copy for d_m_boundary_dof_z from h_m_boundary_dof_z with size equal to the size of the m_mapBoundaryDofIds map and data type int
    checkCudaErrors(hipMemcpy(d_m_boundary_dof_z, h_m_boundary_dof_z, size_of_boundary_dof * sizeof(int), hipMemcpyHostToDevice));

    // --  PARTICLE LEVEL STATISTICAL VARIABLES -- //

    // Allocate memory for the d_m_is_escaped_particle array, size N_Particles and data type int
    checkCudaErrors(hipMalloc((void**)&d_m_is_escaped_particle, N_Particles * sizeof(int)));
    
    // Allocate memory for the d_m_is_deposited_particle array, size N_Particles and data type int
    checkCudaErrors(hipMalloc((void**)&d_m_is_deposited_particle, N_Particles * sizeof(int)));

    // Allocate memory for the d_m_is_error_particle array, size N_Particles and data type int
    checkCudaErrors(hipMalloc((void**)&d_m_is_error_particle, N_Particles * sizeof(int)));

    // Allocate memory for the d_m_is_stagnant_particle array, size N_Particles and data type int
    checkCudaErrors(hipMalloc((void**)&d_m_is_stagnant_particle, N_Particles * sizeof(int)));

    // Allocate memory for the d_m_ghost_particle array, size N_Particles and data type int
    checkCudaErrors(hipMalloc((void**)&d_m_is_ghost_particle, N_Particles * sizeof(int)));


    // Copy the values from the host to the device

    // copy the isParticleDeposited vector to d_m_is_deposited_particle with size N_Particles and data type int
    checkCudaErrors(hipMemcpy(d_m_is_deposited_particle, isParticleDeposited.data(), N_Particles * sizeof(int), hipMemcpyHostToDevice));

    // copy the isEscapedParticle vector to d_m_is_escaped_particle with size N_Particles and data type int
    checkCudaErrors(hipMemcpy(d_m_is_escaped_particle, isEscapedParticle.data(), N_Particles * sizeof(int), hipMemcpyHostToDevice));

    // copy the isStagnantParticle vector to d_m_is_stagnant_particle with size N_Particles and data type int
    checkCudaErrors(hipMemcpy(d_m_is_stagnant_particle, isStagnantParticle.data(), N_Particles * sizeof(int), hipMemcpyHostToDevice));

    // initialize the d_m_is_ghost_particle array with 0 on the device  // There is no Ghost particle array in the given CPU code
    checkCudaErrors(hipMemset(d_m_is_ghost_particle, 0, N_Particles * sizeof(int)));

    
    // -- Allocate memory for the row and column indices of the adjacency matrix -- //
    checkCudaErrors(hipMalloc((void**)&d_m_row_pointer, row_pointer.size() * sizeof(int)));
    checkCudaErrors(hipMalloc((void**)&d_m_col_index, col_index.size() * sizeof(int)));

    // -- Copy the row and column indices of the adjacency matrix -- //
    checkCudaErrors(hipMemcpy(d_m_row_pointer, row_pointer.data(), row_pointer.size() * sizeof(int), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_m_col_index, col_index.data(), col_index.size() * sizeof(int), hipMemcpyHostToDevice));

    cout << "[INFORMATION] Memory allocation and data transfer to GPU is done" << endl;

}

// Setup function to transfer velocity data at every time step
void TParticles::SetupVelocityValues(double *velocity_x_data,
																		 double *velocity_y_data,
																		 double *velocity_z_data,
																		 int N_particles_released,
																		 int N_DOF)
{
    // Copy the velocity values
    checkCudaErrors(hipMemcpy(d_m_velocity_nodal_values_x, velocity_x_data, N_DOF * sizeof(double), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_m_velocity_nodal_values_y, velocity_y_data, N_DOF * sizeof(double), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_m_velocity_nodal_values_z, velocity_z_data, N_DOF * sizeof(double), hipMemcpyHostToDevice));
}

void TParticles::CD_CC_Cuda()
{
    cout << "Inside CD_CC_Cuda" << endl;
    exit(0);
}

// Host wrapper for performing the velocity interpolation at every time step
void TParticles::InterpolateVelocityHostWrapper(double time_step,int N_Particles_released,int N_DOF,int N_Cells)
{
    int MAX_THREAD_PER_BLOCK =32;
    int N_threads; 

    if(N_Particles_released >= MAX_THREAD_PER_BLOCK) 
        N_threads = MAX_THREAD_PER_BLOCK;
        
    else
        N_threads = N_Particles_released;
    
    int C_NUM_BLOCKS = std::ceil(double(N_Particles_released)/MAX_THREAD_PER_BLOCK);


   dim3 dimGrid(C_NUM_BLOCKS);
   dim3 dimBlock(N_threads);

   // time the kernel
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventRecord(start, 0);

    Interpolate_Velocity_CUDA<<<dimGrid,dimBlock>>>(// cell Vertices
                                                    d_m_cell_vertices_x,
                                                    d_m_cell_vertices_y,
                                                    d_m_cell_vertices_z,
                                                    // Velocity Nodal Values
                                                    d_m_velocity_nodal_values_x,
                                                    d_m_velocity_nodal_values_y,
                                                    d_m_velocity_nodal_values_z,
                                                    // Particle Positions, Current & previous
                                                    d_m_particle_position_x,
                                                    d_m_particle_position_y,
                                                    d_m_particle_position_z,
                                                    d_m_particle_previous_position_x,
                                                    d_m_particle_previous_position_y,
                                                    d_m_particle_previous_position_z,
                                                    // Particle Velocities, Current and Previous
                                                    d_m_particle_velocity_x,
                                                    d_m_particle_velocity_y,
                                                    d_m_particle_velocity_z,
                                                    d_m_particle_previous_velocity_x,
                                                    d_m_particle_previous_velocity_y,
                                                    d_m_particle_previous_velocity_z,
                                                    // Particle & fluid Parameters
                                                    d_m_particle_density,
                                                    d_m_particle_diameter,
                                                    d_m_fluid_density,
                                                    d_m_dynamic_viscosity_fluid,
                                                    d_m_lambda,
                                                    d_m_gravity_x,
                                                    d_m_gravity_y,
                                                    d_m_gravity_z,
                                                    //Computational Parameters
                                                    d_m_current_cell,
                                                    d_m_previous_cell,
                                                    d_m_global_dof_indices,
                                                    d_m_begin_indices,

                                                    // FEM parameters for deposition
                                                    d_m_is_boundary_cell,
                                                    d_m_corner_id,
                                                    d_m_is_boundary_dof_present,
                                                    d_m_boundary_dof_x,
                                                    d_m_boundary_dof_y,
                                                    d_m_boundary_dof_z,
                                                    d_m_joint_id,
                                                    d_m_joint_normal_x,
                                                    d_m_joint_normal_y,
                                                    d_m_joint_normal_z,
                                                    d_m_joint_coordinate_x,
                                                    d_m_joint_coordinate_y,
                                                    d_m_joint_coordinate_z,

                                                    // STATISTICAL Parameters
                                                    d_m_is_deposited_particle,
                                                    d_m_is_escaped_particle,
                                                    d_m_is_error_particle,
                                                    d_m_is_stagnant_particle,
                                                    d_m_is_ghost_particle,
                                                    d_m_row_pointer,
                                                    d_m_col_index,
                                                    N_Cells,
                                                    N_DOF,
                                                    N_Particles_released,
                                                    time_step);

    hipDeviceSynchronize();

    hipEventCreate(&stop);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop);
    cout << "[INFORMATION] Time taken for the kernel to execute : " << elapsedTime << " ms" << endl;

    // record the time taken for the transfer of data from device to host
    hipEvent_t start1, stop1;
    hipEventCreate(&start1);
    hipEventRecord(start1, 0);


    // Lets transfer the position values back to the host
    checkCudaErrors(hipMemcpy(position_X.data(), d_m_particle_position_x, N_Particles * sizeof(double), hipMemcpyDeviceToHost));
    checkCudaErrors(hipMemcpy(position_Y.data(), d_m_particle_position_y, N_Particles * sizeof(double), hipMemcpyDeviceToHost));
    checkCudaErrors(hipMemcpy(position_Z.data(), d_m_particle_position_z, N_Particles * sizeof(double), hipMemcpyDeviceToHost));

    // Lets transfer the previous position values back to the host
    checkCudaErrors(hipMemcpy(position_X_old.data(), d_m_particle_previous_position_x, N_Particles * sizeof(double), hipMemcpyDeviceToHost));
    checkCudaErrors(hipMemcpy(position_Y_old.data(), d_m_particle_previous_position_y, N_Particles * sizeof(double), hipMemcpyDeviceToHost));
    checkCudaErrors(hipMemcpy(position_Z_old.data(), d_m_particle_previous_position_z, N_Particles * sizeof(double), hipMemcpyDeviceToHost));

    // Lets transfer the velocity values back to the host
    checkCudaErrors(hipMemcpy(velocityX.data(), d_m_particle_velocity_x, N_Particles * sizeof(double), hipMemcpyDeviceToHost));
    checkCudaErrors(hipMemcpy(velocityY.data(), d_m_particle_velocity_y, N_Particles * sizeof(double), hipMemcpyDeviceToHost));
    checkCudaErrors(hipMemcpy(velocityZ.data(), d_m_particle_velocity_z, N_Particles * sizeof(double), hipMemcpyDeviceToHost));

    // transfer previous velocity values back to the host
    checkCudaErrors(hipMemcpy(velocityX_old.data(), d_m_particle_previous_velocity_x, N_Particles * sizeof(double), hipMemcpyDeviceToHost));
    checkCudaErrors(hipMemcpy(velocityY_old.data(), d_m_particle_previous_velocity_y, N_Particles * sizeof(double), hipMemcpyDeviceToHost));
    checkCudaErrors(hipMemcpy(velocityZ_old.data(), d_m_particle_previous_velocity_z, N_Particles * sizeof(double), hipMemcpyDeviceToHost));

    // Transfer the current cell and previous cell back to the host
    checkCudaErrors(hipMemcpy(currentCell.data(), d_m_current_cell, N_Particles * sizeof(int), hipMemcpyDeviceToHost));
    checkCudaErrors(hipMemcpy(previousCell.data(), d_m_previous_cell, N_Particles * sizeof(int), hipMemcpyDeviceToHost));

    // Transfer Statistical variables back to the host
    checkCudaErrors(hipMemcpy(isParticleDeposited.data(), d_m_is_deposited_particle, N_Particles * sizeof(int), hipMemcpyDeviceToHost));
    checkCudaErrors(hipMemcpy(isEscapedParticle.data(), d_m_is_escaped_particle, N_Particles * sizeof(int), hipMemcpyDeviceToHost));
    checkCudaErrors(hipMemcpy(isStagnantParticle.data(), d_m_is_stagnant_particle, N_Particles * sizeof(int), hipMemcpyDeviceToHost));
    checkCudaErrors(hipMemcpy(isErrorParticle.data(), d_m_is_error_particle, N_Particles * sizeof(int), hipMemcpyDeviceToHost));
    checkCudaErrors(hipMemcpy(isGhostParticle.data(), d_m_is_ghost_particle, N_Particles * sizeof(int), hipMemcpyDeviceToHost));

    // record the time taken for the transfer of data from device to host
    hipEventCreate(&stop1);
    hipEventRecord(stop1, 0);
    hipEventSynchronize(stop1);

    float elapsedTime1;
    hipEventElapsedTime(&elapsedTime1, start1, stop1);
    cout << "[INFORMATION] Time taken for the data transfer from device to host : " << elapsedTime1 << " ms" << endl;

}

// Host wrapper for detecting stagnant particles
void TParticles::DetectStagnantParticlesHostWrapper(int N_Particles_released)
{
    int MAX_THREAD_PER_BLOCK = 128;
    int N_threads; 

    if(N_Particles_released >= MAX_THREAD_PER_BLOCK) 
        N_threads = MAX_THREAD_PER_BLOCK;
    else
        N_threads = N_Particles_released;
    
    int C_NUM_BLOCKS = std::ceil(double(N_Particles_released)/MAX_THREAD_PER_BLOCK);


   dim3 dimGrid(C_NUM_BLOCKS);
   dim3 dimBlock(N_threads);

   // time the kernel
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventRecord(start, 0);

    DetectStagnantParticles_CUDA<<<dimGrid,dimBlock>>>(d_m_particle_position_x,
																											 d_m_particle_position_y,
																											 d_m_particle_position_z,
																											 d_m_particle_stagnant_position_x,
																											 d_m_particle_stagnant_position_y,
																											 d_m_particle_stagnant_position_z,
																											 d_m_is_stagnant_particle,
																											 d_m_is_deposited_particle);

    hipDeviceSynchronize();

    hipEventCreate(&stop);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop);
    cout << "[INFORMATION] Time taken for the kernel to detect stagnant particles : " << elapsedTime << " ms" << endl;

    // record the time taken for the transfer of data from device to host
    hipEvent_t start1, stop1;
    hipEventCreate(&start1);
    hipEventRecord(start1, 0);

    // Lets transfer the previous position values back to the host
    checkCudaErrors(hipMemcpy(previousPosition_X.data(), d_m_particle_stagnant_position_x, N_Particles * sizeof(double), hipMemcpyDeviceToHost));
    checkCudaErrors(hipMemcpy(previousPosition_Y.data(), d_m_particle_stagnant_position_y, N_Particles * sizeof(double), hipMemcpyDeviceToHost));
    checkCudaErrors(hipMemcpy(previousPosition_Z.data(), d_m_particle_stagnant_position_z, N_Particles * sizeof(double), hipMemcpyDeviceToHost));

    // Transfer Statistical variables back to the host
    checkCudaErrors(hipMemcpy(isParticleDeposited.data(), d_m_is_deposited_particle, N_Particles * sizeof(int), hipMemcpyDeviceToHost));
    checkCudaErrors(hipMemcpy(isStagnantParticle.data(), d_m_is_stagnant_particle, N_Particles * sizeof(int), hipMemcpyDeviceToHost));

    // record the time taken for the transfer of data from device to host
    hipEventCreate(&stop1);
    hipEventRecord(stop1, 0);
    hipEventSynchronize(stop1);

    float elapsedTime1;
    hipEventElapsedTime(&elapsedTime1, start1, stop1);
    cout << "[INFORMATION] Time taken for the data transfer(stagnant) from device to host : " << elapsedTime1 << " ms" << endl;

}



